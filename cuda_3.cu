
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <iostream>

__global__ void vecAdd(double* A, double* B, double* C, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < n) {
    if (idx % 2 == 0) {
      for (int i = 0; i < idx; ++i) {
        C[idx] += A[idx] + B[idx];
      }
    } else {
      C[idx] = 0.0;
      while (abs(C[idx]) < 1) {
        C[idx] += A[idx] * B[idx];
      }
    }
  }
}

int main(int argc, char** argv) {
  if (argc != 2) {
    std::cerr << "Wrong arguments" << std::endl;
    return 1;
  }
  int n = atoi(argv[1]);
  double *h_a, *h_b, *h_c;
  size_t bytes = n * sizeof(double);

  h_a = (double*)malloc(bytes);
  h_b = (double*)malloc(bytes);
  h_c = (double*)malloc(bytes);

  for (int i = 0; i < n; ++i) {
    h_a[i] = sin(i) * sin(i);
    h_b[i] = cos(i) * cos(i);
  }

  double *d_a, *d_b, *d_c;

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  int block_size, grid_size;
  block_size = 1024;
  grid_size = (n - 1) / block_size + 1;

  hipEvent_t start_gpu, stop_gpu;
  hipEventCreate(&start_gpu);
  hipEventCreate(&stop_gpu);

  hipEventRecord(start_gpu);

  vecAdd<<<grid_size, block_size>>>(d_a, d_b, d_c, n);

  hipDeviceSynchronize();
  hipEventRecord(stop_gpu);

  float delta = 0.0;
  hipEventElapsedTime(&delta, start_gpu, stop_gpu);

  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  for (int i = 0; i < n; ++i) {
    std::cout << h_c[i] << std::endl;
  }
  
  for (int i = 0; i < n; ++i){
          if (h_c[i] != h_a[i] + h_b[i]){
                  std::cout << "Not equal" << std::endl;
                  break;
          }
          if (i == n-1){
                  std::cout << "Equal" << std::endl;
          }
  }

  std::cout << "Elapsed time" << delta << std::endl;

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}
